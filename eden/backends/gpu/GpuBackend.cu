#include "hip/hip_runtime.h"
//
// Created by max on 12-10-21.
//

#include "GpuBackend.h"

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, " CUDA Error %s at line %d in file %s\n",	    \
             hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("CUDA Error %s at line %d in file %s\n",		            \
			 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }


void GpuBackend::execute_work_gpu(EngineConfig &engine_config, SimulatorConfig &config, int step, double time) {
    const float dt = engine_config.dt;

    for (size_t idx = 0; idx < tabs.consecutive_kernels.size(); idx++) {

        if(config.debug){
            printf("consecutive items %lld start\n", (long long)idx);
            // if(my_mpi.rank != 0) continue;
            // continue;
            fflush(stdout);
        }

        RawTables::ConsecutiveIterationCallbacks & cic = tabs.consecutive_kernels.at(idx);
        ((GPUIterationCallback)cic.callback) (
                          cic.start_item,
                          cic.n_items,
                          time,
                          dt,
                          m_global_constants,
                          m_global_const_f32_index,
                          m_global_tables_const_f32_sizes,
                          m_global_tables_const_f32_arrays,
                          m_global_table_const_f32_index,
                          m_global_tables_const_i64_sizes,
                          m_global_tables_const_i64_arrays,
                          m_global_table_const_i64_index,
                          m_global_tables_state_f32_sizes,
                          m_global_tables_stateNow_f32,
                          m_global_tables_stateNext_f32,
                          m_global_table_state_f32_index,
                          m_global_tables_state_i64_sizes,
                          m_global_tables_stateNow_i64,
                          m_global_tables_stateNext_i64,
                          m_global_table_state_i64_index,
                          m_global_state_now,
                          m_global_state_next,
                          m_global_state_f32_index,
                          step
            );

        if(config.debug){
            printf("consecutive items %lld end\n", (long long)idx);
            fflush(stdout);
        }
    }

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    return;
}

bool GpuBackend::copy_data_to_device() {
    CUDA_CHECK_RETURN(hipMalloc(&m_global_constants,                       tabs.global_constants.size()*sizeof(tabs.global_constants[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_const_f32_index,                 tabs.global_constants.size() * sizeof(tabs.global_constants[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_const_f32_index,           tabs.global_const_f32_index.size() * sizeof(tabs.global_const_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_const_i64_index,           tabs.global_table_const_f32_index.size() * sizeof(tabs.global_table_const_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_state_f32_index,           tabs.global_table_const_i64_index.size() * sizeof(tabs.global_table_const_i64_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_state_i64_index,           tabs.global_table_state_f32_index.size() * sizeof(tabs.global_table_state_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_state_f32_index,                 tabs.global_table_state_i64_index.size() * sizeof(tabs.global_table_state_i64_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_state_now,                       tabs.global_state_f32_index.size() * sizeof(tabs.global_state_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_state_next,                      state->state_one.size() * sizeof(state->state_one[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNow_f32,             state->state_two.size() * sizeof(state->state_two[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNow_i64,             state->global_tables_stateOne_f32_arrays.size() * sizeof(state->global_tables_stateOne_f32_arrays[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNext_f32,            state->global_tables_stateOne_i64_arrays.size() * sizeof(state->global_tables_stateOne_i64_arrays[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNext_i64,            state->global_tables_stateTwo_f32_arrays.size() * sizeof(state->global_tables_stateTwo_f32_arrays[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_f32_arrays,         state->global_tables_stateTwo_i64_arrays.size() * sizeof(state->global_tables_stateTwo_i64_arrays[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_i64_arrays,         state->global_tables_const_f32_arrays.size() * sizeof(state->global_tables_const_f32_arrays[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_f32_sizes,          state->global_tables_const_i64_arrays.size() * sizeof(state->global_tables_const_i64_arrays[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_i64_sizes,          state->global_tables_const_f32_sizes.size() * sizeof(state->global_tables_const_f32_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_state_f32_sizes,          state->global_tables_const_i64_sizes.size() * sizeof(state->global_tables_const_i64_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_state_i64_sizes,          state->global_tables_state_f32_sizes.size() * sizeof(state->global_tables_state_f32_sizes[0])));

    CUDA_CHECK_RETURN(hipMemcpy(m_global_constants                 ,  tabs.global_constants.data()                     ,tabs.global_constants.size()*sizeof(tabs.global_constants[0])                                         ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_const_f32_index           ,  tabs.global_const_f32_index.data()               ,tabs.global_constants.size() * sizeof(tabs.global_constants[0])                                       ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_const_f32_index     ,  tabs.global_table_const_f32_index.data()         ,tabs.global_const_f32_index.size() * sizeof(tabs.global_const_f32_index[0])                           ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_const_i64_index     ,  tabs.global_table_const_i64_index.data()         ,tabs.global_table_const_f32_index.size() * sizeof(tabs.global_table_const_f32_index[0])               ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_state_f32_index     ,  tabs.global_table_state_f32_index.data()         ,tabs.global_table_const_i64_index.size() * sizeof(tabs.global_table_const_i64_index[0])               ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_state_i64_index     ,  tabs.global_table_state_i64_index.data()         ,tabs.global_table_state_f32_index.size() * sizeof(tabs.global_table_state_f32_index[0])               ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_state_f32_index           ,  tabs.global_state_f32_index.data()               ,tabs.global_table_state_i64_index.size() * sizeof(tabs.global_table_state_i64_index[0])               ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_state_now                 ,  state->state_one.data()                          ,tabs.global_state_f32_index.size() * sizeof(tabs.global_state_f32_index[0])                           ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_state_next                ,  state->state_two.data()                          ,state->state_one.size() * sizeof(state->state_one[0])                                                 ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNow_f32       ,  state->global_tables_stateOne_f32_arrays.data()  ,state->state_two.size() * sizeof(state->state_two[0])                                                 ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNow_i64       ,  state->global_tables_stateOne_i64_arrays.data()  ,state->global_tables_stateOne_f32_arrays.size() * sizeof(state->global_tables_stateOne_f32_arrays[0]) ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNext_f32      ,  state->global_tables_stateTwo_f32_arrays.data()  ,state->global_tables_stateOne_i64_arrays.size() * sizeof(state->global_tables_stateOne_i64_arrays[0]) ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNext_i64      ,  state->global_tables_stateTwo_i64_arrays.data()  ,state->global_tables_stateTwo_f32_arrays.size() * sizeof(state->global_tables_stateTwo_f32_arrays[0]) ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_f32_arrays   ,  state->global_tables_const_f32_arrays.data()     ,state->global_tables_stateTwo_i64_arrays.size() * sizeof(state->global_tables_stateTwo_i64_arrays[0]) ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_i64_arrays   ,  state->global_tables_const_i64_arrays.data()     ,state->global_tables_const_f32_arrays.size() * sizeof(state->global_tables_const_f32_arrays[0])       ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_f32_sizes    ,  state->global_tables_const_f32_sizes.data()      ,state->global_tables_const_i64_arrays.size() * sizeof(state->global_tables_const_i64_arrays[0])       ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_i64_sizes    ,  state->global_tables_const_i64_sizes.data()      ,state->global_tables_const_f32_sizes.size() * sizeof(state->global_tables_const_f32_sizes[0])         ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_state_f32_sizes    ,  state->global_tables_state_f32_sizes.data()      ,state->global_tables_const_i64_sizes.size() * sizeof(state->global_tables_const_i64_sizes[0])         ,hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_state_i64_sizes    ,  state->global_tables_state_i64_sizes.data()      ,state->global_tables_state_f32_sizes.size() * sizeof(state->global_tables_state_f32_sizes[0])         ,hipMemcpyHostToDevice));
    return true;
}