#include "hip/hip_runtime.h"
//
// Created by max on 12-10-21.
//

#include "GpuBackend.h"

//change the iteration Callback to also contain some GPU specifics.
extern "C" {
typedef void ( *GPUIterationCallback )(
        long long start, long long n_items,
        double time, float dt, const float *__restrict__ global_constants, const long long *__restrict__ /*XXX*/ global_const_f32_index,
        const long long *__restrict__ global_const_table_f32_sizes, const Table_F32 *__restrict__ global_const_table_f32_arrays, long long *__restrict__ /*XXX*/ global_table_const_f32_index,
        const long long *__restrict__ global_const_table_i64_sizes, const Table_I64 *__restrict__ global_const_table_i64_arrays, long long *__restrict__ /*XXX*/ global_table_const_i64_index,
        const long long *__restrict__ global_state_table_f32_sizes, const Table_F32 *__restrict__ global_state_table_f32_arrays, Table_F32 *__restrict__ global_stateNext_table_f32_arrays,
        long long *__restrict__ /*XXX*/ global_table_state_f32_index,
        const long long *__restrict__ global_state_table_i64_sizes, Table_I64 *__restrict__ global_state_table_i64_arrays, Table_I64 *__restrict__ global_stateNext_table_i64_arrays,
        long long *__restrict__ /*XXX*/ global_table_state_i64_index,
        const float *__restrict__ global_state, float *__restrict__ global_stateNext, long long *__restrict__ global_state_f32_index,
        long long step, int threads_per_block, hipStream_t *streams_calculate);
}

//Todo find a way to pass the stream to the calculate kernel
hipStream_t streams_copy;
hipStream_t streams_calculate;

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, " CUDA Error %s at line %d in file %s\n",	    \
             hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("CUDA Error %s at line %d in file %s\n",		            \
			 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }

void GpuBackend::synchronize_gpu() {
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}
void GpuBackend::gpu_init(){

    //create the Statebuffers
    state = new StateBuffers(tabs);

    //create the copy back host pointers
    m_host_state_now               = state->state_one.data();
    m_host_state_next              = state->state_two.data();
    m_host_tables_stateNow_f32     = state->global_tables_stateOne_f32_arrays.data();
    m_host_tables_stateNow_i64     = state->global_tables_stateOne_i64_arrays.data();
    m_host_tables_stateNext_f32    = state->global_tables_stateTwo_f32_arrays.data();
    m_host_tables_stateNext_i64    = state->global_tables_stateTwo_i64_arrays.data();
    m_host_tables_state_f32_sizes  = state->global_tables_state_f32_sizes.data();
    m_host_tables_state_i64_sizes  = state->global_tables_state_i64_sizes.data();

    m_print_state_now              = state->state_print.data();
    m_print_tables_stateNow_f32    = state->global_tables_statePrint_f32_arrays.data();

    //Create the Streams
    CUDA_CHECK_RETURN(hipStreamCreate(&streams_copy));
    CUDA_CHECK_RETURN(hipStreamCreate(&streams_calculate));

    // alloc simple
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_constants, tabs.global_constants.size()*sizeof(tabs.global_constants[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_const_f32_index, tabs.global_const_f32_index.size()*sizeof(tabs.global_const_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_table_const_f32_index, tabs.global_table_const_f32_index.size()*sizeof(tabs.global_table_const_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_table_const_i64_index, tabs.global_table_const_i64_index.size()*sizeof(tabs.global_table_const_i64_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_table_state_f32_index, tabs.global_table_state_f32_index.size()*sizeof(tabs.global_table_state_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_table_state_i64_index, tabs.global_table_state_i64_index.size()*sizeof(tabs.global_table_state_i64_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_state_f32_index, tabs.global_state_f32_index.size()*sizeof(tabs.global_state_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_state_now, state->state_one.size()*sizeof(state->state_one[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_state_next, state->state_two.size()*sizeof(state->state_two[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_const_f32_sizes, state->global_tables_const_f32_sizes.size()*sizeof(state->global_tables_const_f32_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_const_i64_sizes, state->global_tables_const_i64_sizes.size()*sizeof(state->global_tables_const_i64_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_state_f32_sizes, state->global_tables_state_f32_sizes.size()*sizeof(state->global_tables_state_f32_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_state_i64_sizes, state->global_tables_state_i64_sizes.size()*sizeof(state->global_tables_state_i64_sizes[0])));

    // copy simple
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_constants,                tabs.global_constants.data(),                   tabs.global_constants.size()*sizeof(tabs.global_constants[0]),                                  hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_const_f32_index,          tabs.global_const_f32_index.data(),             tabs.global_const_f32_index.size()*sizeof(tabs.global_const_f32_index[0]),                      hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_table_const_f32_index,    tabs.global_table_const_f32_index.data(),       tabs.global_table_const_f32_index.size()*sizeof(tabs.global_table_const_f32_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_table_const_i64_index,    tabs.global_table_const_i64_index.data(),       tabs.global_table_const_i64_index.size()*sizeof(tabs.global_table_const_i64_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_table_state_f32_index,    tabs.global_table_state_f32_index.data(),       tabs.global_table_state_f32_index.size()*sizeof(tabs.global_table_state_f32_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_table_state_i64_index,    tabs.global_table_state_i64_index.data(),       tabs.global_table_state_i64_index.size()*sizeof(tabs.global_table_state_i64_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_state_f32_index,          tabs.global_state_f32_index.data(),             tabs.global_state_f32_index.size()*sizeof(tabs.global_state_f32_index[0]),                      hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_state_now,                state->state_one.data(),                        state->state_one.size()*sizeof(state->state_one[0]),                                            hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_state_next,               state->state_two.data(),                        state->state_two.size()*sizeof(state->state_two[0]),                                            hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_const_f32_sizes,   state->global_tables_const_f32_sizes.data(),    state->global_tables_const_f32_sizes.size()*sizeof(state->global_tables_const_f32_sizes[0]),    hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_const_i64_sizes,   state->global_tables_const_i64_sizes.data(),    state->global_tables_const_i64_sizes.size()*sizeof(state->global_tables_const_i64_sizes[0]),    hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_state_f32_sizes,   state->global_tables_state_f32_sizes.data(),    state->global_tables_state_f32_sizes.size()*sizeof(state->global_tables_state_f32_sizes[0]),    hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_state_i64_sizes,   state->global_tables_state_i64_sizes.data(),    state->global_tables_state_i64_sizes.size()*sizeof(state->global_tables_state_i64_sizes[0]),    hipMemcpyHostToDevice));

    /* double pointers */
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_stateNow_f32, state->global_tables_stateOne_f32_arrays.size()*sizeof(state->global_tables_stateOne_f32_arrays[0]))); // state->global_tables_state_f32_sizes.data()
    std::vector<float*> temp_f32;
    for (size_t i = 0; i < state->global_tables_stateOne_f32_arrays.size(); i++) {
        size_t size = state->global_tables_state_f32_sizes[i];
        float * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateOne_f32_arrays[i], size*sizeof(float), hipMemcpyHostToDevice));
        temp_f32.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_stateNow_f32, temp_f32.data(), temp_f32.size()*sizeof(float*), hipMemcpyHostToDevice));
    temp_f32.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_stateNext_f32, state->global_tables_stateOne_f32_arrays.size()*sizeof(state->global_tables_stateOne_f32_arrays[0]))); // state->global_tables_state_f32_sizes.data()
    for (size_t i = 0; i < state->global_tables_stateOne_f32_arrays.size(); i++) {
        size_t size = state->global_tables_state_f32_sizes[i];
        float * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateOne_f32_arrays[i], size*sizeof(float), hipMemcpyHostToDevice));
        temp_f32.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_stateNext_f32, temp_f32.data(), temp_f32.size()*sizeof(float*), hipMemcpyHostToDevice));
    temp_f32.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_const_f32_arrays, state->global_tables_const_f32_arrays.size()*sizeof(state->global_tables_const_f32_arrays[0]))); // state->global_tables_const_f32_sizes.data()
    for (size_t i = 0; i < state->global_tables_const_f32_arrays.size(); i++) {
        size_t size = state->global_tables_const_f32_sizes[i];
        float * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_const_f32_arrays[i], size*sizeof(float), hipMemcpyHostToDevice));
        temp_f32.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_const_f32_arrays, temp_f32.data(), temp_f32.size()*sizeof(float*), hipMemcpyHostToDevice));
    temp_f32.clear();

    std::vector<long long*> temp_i64;
    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_stateNow_i64, state->global_tables_stateOne_i64_arrays.size()*sizeof(state->global_tables_stateOne_i64_arrays[0]))); // state->global_tables_state_i64_sizes.data(),
    for (size_t i = 0; i < state->global_tables_stateOne_i64_arrays.size(); i++) {
        size_t size = state->global_tables_state_i64_sizes[i];
        long long * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(long long)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateOne_i64_arrays[i], size*sizeof(long long), hipMemcpyHostToDevice));
        temp_i64.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_stateNow_i64, temp_i64.data(), temp_i64.size()*sizeof(long long*), hipMemcpyHostToDevice));
    temp_i64.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_stateNext_i64, state->global_tables_stateTwo_i64_arrays.size()*sizeof(state->global_tables_stateTwo_i64_arrays[0]))); // state->global_tables_state_i64_sizes.data(),
    for (size_t i = 0; i < state->global_tables_stateTwo_i64_arrays.size(); i++) {
        size_t size = state->global_tables_state_i64_sizes[i];
        long long * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(long long)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateTwo_i64_arrays[i], size*sizeof(long long), hipMemcpyHostToDevice));
        temp_i64.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_stateNext_i64, temp_i64.data(), temp_i64.size()*sizeof(long long*), hipMemcpyHostToDevice));
    temp_i64.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_gpu_tables_const_i64_arrays, state->global_tables_const_i64_arrays.size()*sizeof(state->global_tables_const_i64_arrays[0]))); // state->global_tables_state_i64_sizes.data()
    for (size_t i = 0; i < state->global_tables_const_i64_arrays.size(); i++) {
        size_t size = state->global_tables_const_i64_sizes[i];
        long long * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(long long)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_const_i64_arrays[i], size*sizeof(long long), hipMemcpyHostToDevice));
        temp_i64.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_gpu_tables_const_i64_arrays, temp_i64.data(), temp_i64.size()*sizeof(long long*), hipMemcpyHostToDevice));
    temp_i64.clear();
}


void GpuBackend::execute_work_gpu(EngineConfig &engine_config, SimulatorConfig &config, int step, double time, int threads_per_block) {

    //Start by updating the MPI buffers if neccesary
#ifdef USE_MPI
    //only needed if there are potentially incoming messages.
//    if(engine_config.my_mpi.world_size > 1) {
    //TABLE_F32

        std::vector<Table_F32> temp32b(state->global_tables_stateTwo_f32_arrays.size(), nullptr);
        CUDA_CHECK_RETURN(hipMemcpy( temp32b.data(),
                                      m_gpu_tables_stateNow_f32,
                                      temp32b.size()*sizeof(Table_F32),
                                      hipMemcpyDeviceToHost));

        for (size_t i = 0; i < temp32b.size(); i++) {
            size_t size = m_host_tables_state_f32_sizes[i];
            if(config.debug) {
                printf("32 in - pointer %zu %zu %p \n", size, i, temp32b[i]);
            }
            if(size) {
                if(config.debug) {
                    for (size_t j = 0; j < size; j++)
                        printf(" %f\n", m_host_tables_stateNow_f32[i][j]);
                }
                CUDA_CHECK_RETURN(hipMemcpy(
                        temp32b[i],
                        m_host_tables_stateNow_f32[i],
                        size * sizeof(float),
                        hipMemcpyHostToDevice));
            }
        }

        //Table_I64
        std::vector<Table_I64> temp64b(state->global_tables_stateTwo_i64_arrays.size(), nullptr);
        CUDA_CHECK_RETURN(hipMemcpy( temp64b.data(),
                                      m_gpu_tables_stateNow_i64,
                                      temp64b.size()*sizeof(Table_I64),
                                      hipMemcpyDeviceToHost));


        for (size_t i = 0; i < temp64b.size(); i++) {
            size_t size = m_host_tables_state_i64_sizes[i];
            if(config.debug) {
                printf("64 in - pointer %zu %zu %p \n", size, i, temp64b[i]);
            }
            if(size) {
                if(config.debug) {
                    for (size_t j = 0; j < size; j++)
                        printf(" %lld\n", m_host_tables_stateNow_i64[i][j]);
                }
                CUDA_CHECK_RETURN(hipMemcpy(
                        temp64b[i],
                        m_host_tables_stateNow_i64[i],
                        size * sizeof(long long int),
                        hipMemcpyHostToDevice));
            }
        }
//    }
#endif

    const float dt = engine_config.dt;
    for (size_t idx = 0; idx < tabs.consecutive_kernels.size(); idx++) {
        RawTables::ConsecutiveIterationCallbacks & cic = tabs.consecutive_kernels.at(idx);
        if(config.debug){
            printf("consecutive item %lld (start %ld length %ld) start\n", (long long)idx, (long)cic.start_item, (long)cic.n_items);
            // if(my_mpi.rank != 0) continue;
            // continue;
            fflush(stdout);
        }
        ((GPUIterationCallback)cic.callback) (
                          cic.start_item,
                          cic.n_items,
                          time,
                          dt,
                          m_gpu_constants,
                          m_gpu_const_f32_index,
                          m_gpu_tables_const_f32_sizes,
                          m_gpu_tables_const_f32_arrays,
                          m_gpu_table_const_f32_index,
                          m_gpu_tables_const_i64_sizes,
                          m_gpu_tables_const_i64_arrays,
                          m_gpu_table_const_i64_index,
                          m_gpu_tables_state_f32_sizes,
                          m_gpu_tables_stateNow_f32,
                          m_gpu_tables_stateNext_f32,
                          m_gpu_table_state_f32_index,
                          m_gpu_tables_state_i64_sizes,
                          m_gpu_tables_stateNow_i64,
                          m_gpu_tables_stateNext_i64,
                          m_gpu_table_state_i64_index,
                          m_gpu_state_now,
                          m_gpu_state_next,
                          m_gpu_state_f32_index,
                          step,
                          threads_per_block,
                          &streams_calculate
            );

        if(config.debug){
            printf("consecutive items %lld end\n", (long long)idx);
            fflush(stdout);
        }
    }

    //copy back for printing of stuff.
    CUDA_CHECK_RETURN(hipMemcpyAsync(
            m_host_state_now,
            m_gpu_state_now,
            state->state_one.size()*sizeof(state->state_one[0]),
            hipMemcpyDeviceToHost,streams_copy));


#ifdef USE_MPI
    //to copy back to the host for MPI communication
    synchronize_gpu();
    //TABLE_F32
    //todo store this vector
    std::vector<Table_F32> temp32(state->global_tables_stateTwo_f32_arrays.size(), nullptr);
    CUDA_CHECK_RETURN(hipMemcpy( temp32.data(),
                                  m_gpu_tables_stateNext_f32,
                                  temp32.size()*sizeof(Table_F32),
                                  hipMemcpyDeviceToHost));

    for (size_t i = 0; i < temp32.size(); i++) {
        size_t size = m_host_tables_state_f32_sizes[i];
        CUDA_CHECK_RETURN(hipMemcpy(
                m_host_tables_stateNext_f32[i],
                temp32[i],
                size*sizeof(float),
                hipMemcpyDeviceToHost));
        if(config.debug) {
            printf("32 - out pointer %zu %zu %p \n", size, i, temp32[i]);
            for (size_t j = 0; j < size; j++)
                printf(" %f\n", m_host_tables_stateNext_f32[i][j]);
        }
    }

    //moet hier na!!
    CUDA_CHECK_RETURN(hipMemcpy(
            m_host_state_next,
            m_gpu_state_next,
            state->state_one.size()*sizeof(state->state_one[0]),
            hipMemcpyDeviceToHost));
    if(config.debug) {
        for (size_t j = 0; j < state->state_one.size(); j++)
            printf(" %f\n", m_host_state_next[j]);
    }

    //Table_I64
    //todo store this vector
    std::vector<Table_I64> temp64(state->global_tables_stateTwo_i64_arrays.size(), nullptr);
    CUDA_CHECK_RETURN(hipMemcpy( temp64.data(),
                                  m_gpu_tables_stateNext_i64,
                                  temp64.size()*sizeof(Table_I64),
                                  hipMemcpyDeviceToHost));

    for (size_t i = 0; i < temp64.size(); i++) {
        size_t size = m_host_tables_state_i64_sizes[i];
        if(config.debug) {
            printf("64 - out pointer %zu %zu %p \n",size,i,temp64[i]);
        }
        if(size) {
            CUDA_CHECK_RETURN(hipMemcpy(
                    m_host_tables_stateNext_i64[i],
                    temp64[i],
                    size * sizeof(long long int),
                    hipMemcpyDeviceToHost));
            if(config.debug) {
                for(size_t j = 0; j <size;j++) {
                    printf(" %lld\n", m_host_tables_stateNext_i64[i][j]);
                }
            }
        }
    }
#endif
}



