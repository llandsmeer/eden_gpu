#include "hip/hip_runtime.h"
//
// Created by max on 12-10-21.
//

#include "GpuBackend.h"

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, " CUDA Error %s at line %d in file %s\n",	    \
             hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("CUDA Error %s at line %d in file %s\n",		            \
			 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }

bool GpuBackend::copy_data_to_device() {

    printf("hello dit werkt cuda \n");
    return true;
}