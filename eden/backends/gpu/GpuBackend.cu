#include "hip/hip_runtime.h"
//
// Created by max on 12-10-21.
//

#include "GpuBackend.h"

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, " CUDA Error %s at line %d in file %s\n",	    \
             hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("CUDA Error %s at line %d in file %s\n",		            \
			 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }


void GpuBackend::execute_work_gpu(EngineConfig &engine_config, SimulatorConfig &config, int step, double time) {
    const float dt = engine_config.dt;

    for (size_t idx = 0; idx < tabs.consecutive_kernels.size(); idx++) {

        if(config.debug){
            printf("consecutive items %lld start\n", (long long)idx);
            // if(my_mpi.rank != 0) continue;
            // continue;
            fflush(stdout);
        }

        RawTables::ConsecutiveIterationCallbacks & cic = tabs.consecutive_kernels.at(idx);
        ((GPUIterationCallback)cic.callback) (
                          cic.start_item,
                          cic.n_items,
                          time,
                          dt,
                          m_global_constants,
                          m_global_const_f32_index,
                          m_global_tables_const_f32_sizes,
                          m_global_tables_const_f32_arrays,
                          m_global_table_const_f32_index,
                          m_global_tables_const_i64_sizes,
                          m_global_tables_const_i64_arrays,
                          m_global_table_const_i64_index,
                          m_global_tables_state_f32_sizes,
                          m_global_tables_stateNow_f32,
                          m_global_tables_stateNext_f32,
                          m_global_table_state_f32_index,
                          m_global_tables_state_i64_sizes,
                          m_global_tables_stateNow_i64,
                          m_global_tables_stateNext_i64,
                          m_global_table_state_i64_index,
                          m_global_state_now,
                          m_global_state_next,
                          m_global_state_f32_index,
                          step
            );

        if(config.debug){
            printf("consecutive items %lld end\n", (long long)idx);
            fflush(stdout);
        }
    }

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    return;
}

bool GpuBackend::copy_data_to_device() {

    // alloc simple
    CUDA_CHECK_RETURN(hipMalloc(&m_global_constants, tabs.global_constants.size()*sizeof(tabs.global_constants[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_const_f32_index, tabs.global_const_f32_index.size()*sizeof(tabs.global_const_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_const_f32_index, tabs.global_table_const_f32_index.size()*sizeof(tabs.global_table_const_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_const_i64_index, tabs.global_table_const_i64_index.size()*sizeof(tabs.global_table_const_i64_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_state_f32_index, tabs.global_table_state_f32_index.size()*sizeof(tabs.global_table_state_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_table_state_i64_index, tabs.global_table_state_i64_index.size()*sizeof(tabs.global_table_state_i64_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_state_f32_index, tabs.global_state_f32_index.size()*sizeof(tabs.global_state_f32_index[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_state_now, state->state_one.size()*sizeof(state->state_one[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_state_next, state->state_two.size()*sizeof(state->state_two[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_f32_sizes, state->global_tables_const_f32_sizes.size()*sizeof(state->global_tables_const_f32_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_i64_sizes, state->global_tables_const_i64_sizes.size()*sizeof(state->global_tables_const_i64_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_state_f32_sizes, state->global_tables_state_f32_sizes.size()*sizeof(state->global_tables_state_f32_sizes[0])));
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_state_i64_sizes, state->global_tables_state_i64_sizes.size()*sizeof(state->global_tables_state_i64_sizes[0])));

    // copy simple
    CUDA_CHECK_RETURN(hipMemcpy(m_global_constants,                tabs.global_constants.data(),                   tabs.global_constants.size()*sizeof(tabs.global_constants[0]),                                  hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_const_f32_index,          tabs.global_const_f32_index.data(),             tabs.global_const_f32_index.size()*sizeof(tabs.global_const_f32_index[0]),                      hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_const_f32_index,    tabs.global_table_const_f32_index.data(),       tabs.global_table_const_f32_index.size()*sizeof(tabs.global_table_const_f32_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_const_i64_index,    tabs.global_table_const_i64_index.data(),       tabs.global_table_const_i64_index.size()*sizeof(tabs.global_table_const_i64_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_state_f32_index,    tabs.global_table_state_f32_index.data(),       tabs.global_table_state_f32_index.size()*sizeof(tabs.global_table_state_f32_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_table_state_i64_index,    tabs.global_table_state_i64_index.data(),       tabs.global_table_state_i64_index.size()*sizeof(tabs.global_table_state_i64_index[0]),          hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_state_f32_index,          tabs.global_state_f32_index.data(),             tabs.global_state_f32_index.size()*sizeof(tabs.global_state_f32_index[0]),                      hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_state_now,                state->state_one.data(),                        state->state_one.size()*sizeof(state->state_one[0]),                                            hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_state_next,               state->state_two.data(),                        state->state_two.size()*sizeof(state->state_two[0]),                                            hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_f32_sizes,   state->global_tables_const_f32_sizes.data(),    state->global_tables_const_f32_sizes.size()*sizeof(state->global_tables_const_f32_sizes[0]),    hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_i64_sizes,   state->global_tables_const_i64_sizes.data(),    state->global_tables_const_i64_sizes.size()*sizeof(state->global_tables_const_i64_sizes[0]),    hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_state_f32_sizes,   state->global_tables_state_f32_sizes.data(),    state->global_tables_state_f32_sizes.size()*sizeof(state->global_tables_state_f32_sizes[0]),    hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_state_i64_sizes,   state->global_tables_state_i64_sizes.data(),    state->global_tables_state_i64_sizes.size()*sizeof(state->global_tables_state_i64_sizes[0]),    hipMemcpyHostToDevice));

    /* double pointers */

    std::vector<float*> temp_f32;

    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNow_f32, state->global_tables_stateOne_f32_arrays.size()*sizeof(state->global_tables_stateOne_f32_arrays[0]))); // state->global_tables_state_f32_sizes.data()
    for (size_t i = 0; i < state->global_tables_stateOne_f32_arrays.size(); i++) {
        size_t size = state->global_tables_state_f32_sizes[i];
        float * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateOne_f32_arrays[i], size*sizeof(float), hipMemcpyHostToDevice));
        temp_f32.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNow_f32, temp_f32.data(), temp_f32.size()*sizeof(float*), hipMemcpyHostToDevice));
    temp_f32.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNext_f32, state->global_tables_stateTwo_f32_arrays.size()*sizeof(state->global_tables_stateTwo_f32_arrays[0]))); // state->global_tables_state_f32_sizes.data()
    for (size_t i = 0; i < state->global_tables_stateTwo_f32_arrays.size(); i++) {
        size_t size = state->global_tables_state_f32_sizes[i];
        float * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateTwo_f32_arrays[i], size*sizeof(float), hipMemcpyHostToDevice));
        temp_f32.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNext_f32, temp_f32.data(), temp_f32.size()*sizeof(float*), hipMemcpyHostToDevice));
    temp_f32.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_f32_arrays, state->global_tables_const_f32_arrays.size()*sizeof(state->global_tables_const_f32_arrays[0]))); // state->global_tables_const_f32_sizes.data()
    for (size_t i = 0; i < state->global_tables_const_f32_arrays.size(); i++) {
        size_t size = state->global_tables_const_f32_sizes[i];
        float * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_const_f32_arrays[i], size*sizeof(float), hipMemcpyHostToDevice));
        temp_f32.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_f32_arrays, temp_f32.data(), temp_f32.size()*sizeof(float*), hipMemcpyHostToDevice));
    temp_f32.clear();

    std::vector<long long*> temp_i64;
    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNow_i64, state->global_tables_stateOne_i64_arrays.size()*sizeof(state->global_tables_stateOne_i64_arrays[0]))); // state->global_tables_state_i64_sizes.data(),
    for (size_t i = 0; i < state->global_tables_stateOne_i64_arrays.size(); i++) {
        size_t size = state->global_tables_state_i64_sizes[i];
        long long * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(long long)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateOne_i64_arrays[i], size*sizeof(long long), hipMemcpyHostToDevice));
        temp_i64.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNow_i64, temp_i64.data(), temp_i64.size()*sizeof(long long*), hipMemcpyHostToDevice));
    temp_i64.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_stateNext_i64, state->global_tables_stateTwo_i64_arrays.size()*sizeof(state->global_tables_stateTwo_i64_arrays[0]))); // state->global_tables_state_i64_sizes.data(),
    for (size_t i = 0; i < state->global_tables_stateTwo_i64_arrays.size(); i++) {
        size_t size = state->global_tables_state_i64_sizes[i];
        long long * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(long long)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_stateTwo_i64_arrays[i], size*sizeof(long long), hipMemcpyHostToDevice));
        temp_i64.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_stateNext_i64, temp_i64.data(), temp_i64.size()*sizeof(long long*), hipMemcpyHostToDevice));
    temp_i64.clear();

    CUDA_CHECK_RETURN(hipMalloc(&m_global_tables_const_i64_arrays, state->global_tables_const_i64_arrays.size()*sizeof(state->global_tables_const_i64_arrays[0]))); // state->global_tables_state_i64_sizes.data()
    for (size_t i = 0; i < state->global_tables_const_i64_arrays.size(); i++) {
        size_t size = state->global_tables_const_i64_sizes[i];
        printf("Allocating subarray %d of size %lld\n", i, (long long)size);
        long long * item_ptr;
        CUDA_CHECK_RETURN(hipMalloc(&item_ptr, size*sizeof(long long)));
        CUDA_CHECK_RETURN(hipMemcpy(item_ptr, state->global_tables_const_i64_arrays[i], size*sizeof(long long), hipMemcpyHostToDevice));
        temp_i64.push_back(item_ptr);
    }
    CUDA_CHECK_RETURN(hipMemcpy(m_global_tables_const_i64_arrays, temp_i64.data(), temp_i64.size()*sizeof(long long*), hipMemcpyHostToDevice));
    temp_i64.clear();

    return true;
}

float * GpuBackend::global_state_now() const {
    CUDA_CHECK_RETURN(hipMemcpy(
                state->state_one.data(),
                m_global_state_now,
                state->state_one.size()*sizeof(state->state_one[0]),
                hipMemcpyDeviceToHost));
    return state->state_one.data();
}

Table_F32 * GpuBackend::global_tables_stateNow_f32 () const {
    // here be dragons
    // XXX TODO: remove temp allocation - we can just keep the temp_f32 vector from allocation
    // XXX TODO: call this function only when using MPI
    // Also, state->global_tables_stateOne_f32_arrays points to state->state_one in some way
    // leading to overwrites in certain cases, but not others (?)
    std::vector<float*> temp(state->global_tables_stateTwo_f32_arrays.size(), 0);
    CUDA_CHECK_RETURN(hipMemcpy(
                temp.data(),
                m_global_tables_stateNow_f32,
                temp.size()*sizeof(float*),
                hipMemcpyDeviceToHost));
    for (size_t i = 0; i < temp.size(); i++) {
        size_t size = state->global_tables_state_f32_sizes[i];
        CUDA_CHECK_RETURN(hipMemcpy(
                    state->global_tables_stateTwo_f32_arrays[i],
                    temp[i],
                    size*sizeof(float),
                    hipMemcpyDeviceToHost));
    }
    return state->global_tables_stateTwo_f32_arrays.data();
}
