#include "hip/hip_runtime.h"
//
// Created by max on 04-10-21.
//

#include <stdio.h>
#include <iostream>

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, " CUDA Error %s at line %d in file %s\n",	    \
             hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("CUDA Error %s at line %d in file %s\n",		            \
			 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }


bool GPU_checker(){
    int Amount_of_GPUS_detected = 0;
    CUDA_CHECK_RETURN(hipGetDeviceCount(&Amount_of_GPUS_detected));
    std::cout << "amount of gpus detected: " << Amount_of_GPUS_detected << std::endl;
    //todo
    hipDeviceProp_t prop;
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, 0));
    std::cout << "      PCI device id:                " << prop.pciBusID << "\n";
    std::cout << "      Device name:                  " << prop.name << "\n";
    std::cout << "      Clock Rate (KHz):             " << prop.clockRate << "\n";
    std::cout << "      Memory Clock Rate (KHz):      " << prop.memoryClockRate << "\n";
    std::cout << "      Memory Bus Width (bits):      " << prop.memoryBusWidth << "\n";
    std::cout << "      Peak Memory Bandwidth (GB/s): " << (2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6) << "\n";
    std::cout << "      Total global memory (Gbytes): " << (prop.totalGlobalMem / 1000000000) << "\n";
    std::cout << "      Compute cabability :          " << prop.major << "." << prop.minor << "\n";
    std::cout << "      Number of multiprocessors :   " << prop.multiProcessorCount << " \n";
    return true;
}

void test(){
    GPU_checker();
}
