#include "hip/hip_runtime.h"
//
// Created by max on 04-10-21.
//

#include "GPU_helpers.h"

#define CUDA_CHECK_RETURN(value) {										\
	hipError_t _m_cudaStat = value;									\
	if (_m_cudaStat != hipSuccess) {									\
		fprintf(stderr, " CUDA Error %s at line %d in file %s\n",	    \
             hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
        printf("CUDA Error %s at line %d in file %s\n",		            \
			 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		if(value == 2) exit(2);                                         \
		exit(1);														\
	} }

bool test_GPU(LogContext &logC){
    INIT_LOG(&logC.log_file,logC.mpi_rank);
    int Amount_of_GPUS_detected = 0;
    CUDA_CHECK_RETURN(hipGetDeviceCount(&Amount_of_GPUS_detected));
    log(LOG_INFO) << "amount of gpus detected: " << Amount_of_GPUS_detected << LOG_ENDL;
    hipDeviceProp_t prop{};
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, 0));
    log(LOG_INFO) << "      PCI device id:                " << prop.pciBusID << LOG_ENDL;
    log(LOG_INFO) << "      Device name:                  " << prop.name << LOG_ENDL;
    log(LOG_INFO) << "      Clock Rate (KHz):             " << prop.clockRate << LOG_ENDL;
    log(LOG_INFO) << "      Memory Clock Rate (KHz):      " << prop.memoryClockRate << LOG_ENDL;
    log(LOG_INFO) << "      Memory Bus Width (bits):      " << prop.memoryBusWidth << LOG_ENDL;
    log(LOG_INFO) << "      Peak Memory Bandwidth (GB/s): " << (2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6) << LOG_ENDL;
    log(LOG_INFO) << "      Total global memory (Gbytes): " << (prop.totalGlobalMem / 1000000000) << LOG_ENDL;
    log(LOG_INFO) << "      Compute cabability :          " << prop.major << "." << prop.minor << LOG_ENDL;
    log(LOG_INFO) << "      Number of multiprocessors :   " << prop.multiProcessorCount << LOG_ENDL;
    return Amount_of_GPUS_detected;
}
